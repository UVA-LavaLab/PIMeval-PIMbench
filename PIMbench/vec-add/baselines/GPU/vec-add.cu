#include "hip/hip_runtime.h"
/* File:     vec-add.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 */

#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <nvml.h>
#include <chrono>
#include <thread>

#include "utilBaselines.h"

vector<float> A;
vector<float> B;
vector<float> C;

using namespace std;

// Struct for Parameters
struct Params
{
    uint64_t vectorSize = 2048; // Default vector size
};

/**
 * @brief Displays usage information
 */
void usage()
{
    cerr << "\nUsage:  ./vec-add.out [options]\n"
         << "\nOptions:\n"
         << "    -l    vector size (default=2048 elements)\n"
         << "    -h    display this help message\n";
}

/**
 * @brief Parses command line input parameters
 * @param argc Number of command line arguments
 * @param argv Array of command line arguments
 * @return Parsed parameters
 */
Params parseParams(int argc, char **argv)
{
    Params params;

    int opt;
    while ((opt = getopt(argc, argv, "l:h")) != -1)
    {
        switch (opt)
        {
        case 'h':
            usage();
            exit(0);
        case 'l':
            params.vectorSize = stoull(optarg);
            break;
        default:
            cerr << "\nUnrecognized option: " << opt << "\n";
            usage();
            exit(1);
        }
    }

    return params;
}

__global__ void vecAdd(float* x, float* y, float* z)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    z[index] = x[index] + y[index];
}

int main(int argc, char *argv[])
{
    // Parse input parameters
    Params params = parseParams(argc, argv);
    uint64_t vectorSize = params.vectorSize;
    float *x, *y, *z;
    int blockSize = 1024;
    u_int64_t numBlock = (vectorSize + blockSize - 1) / blockSize;

    uint64_t n_pad = numBlock * blockSize;

    getVector<float>(n_pad, A);
    getVector<float>(n_pad, B);
    C.resize(n_pad);
    std::cout << "Running vector addition for GPU on vector of size: " << vectorSize << std::endl;

    hipError_t errorCode;

    errorCode = hipMalloc(&x, n_pad * sizeof(float));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }
    errorCode = hipMalloc(&y, n_pad * sizeof(float));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }
    errorCode = hipMalloc(&z, n_pad * sizeof(float));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    errorCode = hipMemcpy(x, A.data(), vectorSize * sizeof(float), hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    errorCode = hipMemcpy(y, B.data(), vectorSize * sizeof(float), hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    std::cout << "Launching CUDA Kernel." << std::endl;

    auto [timeElapsed, avgPower, energy] = measureCUDAPowerAndElapsedTime([&]() {
        vecAdd<<<numBlock, blockSize>>>(x, y, z);
        hipDeviceSynchronize(); // ensure all are done
    });

    // Check for kernel launch errors
    errorCode = hipGetLastError();
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }
    
    printf("\nExecution time of vector addition = %f ms\n", timeElapsed);
    printf("Average Power = %f mW\n", avgPower);
    printf("Energy Consumption = %f mJ\n", energy);

    errorCode = hipMemcpy(C.data(), z, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    for (uint64_t i = 0; i < vectorSize; i++)
    {
        if (C[i] != A[i] + B[i])
        {
            cout << "Addition failed at index: " << i << " value: " << z[i] << endl;
            break;
        }
    }

    /* Free memory */
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
} /* main */
