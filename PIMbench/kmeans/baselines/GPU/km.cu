#include "hip/hip_runtime.h"
/* File:     km.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 */

#include <algorithm>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <stdexcept>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <getopt.h>
#include <assert.h>
#include <stdint.h>
#include <iostream>
#include <unordered_map>
#include <chrono>
#include <vector>
#include <iomanip>
#include <float.h>
#include <limits.h>

#include "utilBaselines.h"

using namespace std;

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t numPoints;
  int maxItr;
  int dimension;
  int k;
  int numThreads;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./km.out [options]"
          "\n"
          "\n    -t    # of threads (default=8)"
          "\n    -p    number of points (default=1024 points)"
          "\n    -k    value of K (default=20)"
          "\n    -d    number of features (default=2 dimensions)"
          "\n    -i    max iteration (default=5 iteration)"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.numPoints = 1024;
  p.k = 20;
  p.dimension = 2;
  p.numThreads = 8;
  p.maxItr = 5;

  int opt;
  while ((opt = getopt(argc, argv, "p:k:d:i:t:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'p':
      p.numPoints = atoll(optarg);
      break;
    case 'k':
      p.k = atoi(optarg);
      break;
    case 'd':
      p.dimension = atoi(optarg);
      break;
    case 't':
      p.numThreads = atoi(optarg);
      break;
    case 'i':
      p.maxItr = atoi(optarg);
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }

  return p;
}

struct Data
{
  Data(int size) : size(size), bytes(size * sizeof(int32_t))
  {
    hipError_t errorCode;
    errorCode = hipMalloc(&x, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMalloc(&y, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMemset(x, 0, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMemset(y, 0, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
  }

  Data(int size, std::vector<int32_t> &h_x, std::vector<int32_t> &h_y)
      : size(size), bytes(size * sizeof(int32_t))
  {
    hipError_t errorCode;
    errorCode = hipMalloc(&x, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMalloc(&y, bytes);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMemcpy(x, h_x.data(), bytes, hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
    errorCode = hipMemcpy(y, h_y.data(), bytes, hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
    } 
  }

  ~Data()
  {
    hipFree(x);
    hipFree(y);
  }

  int32_t *x{nullptr};
  int32_t *y{nullptr};
  int size{0};
  int bytes{0};
};

__device__ int32_t
squared_l2_distance(int32_t x_1, int32_t y_1, int32_t x_2, int32_t y_2)
{
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

__global__ void fine_reduce(const int32_t *__restrict__ data_x,
                            const int32_t *__restrict__ data_y,
                            int data_size,
                            const int32_t *__restrict__ means_x,
                            const int32_t *__restrict__ means_y,
                            int32_t *__restrict__ new_sums_x,
                            int32_t *__restrict__ new_sums_y,
                            int k,
                            int *__restrict__ counts)
{
  extern __shared__ int32_t shared_data[];

  const int local_index = threadIdx.x;
  const int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_index >= data_size)
    return;

  // Load the mean values into shared memory.
  if (local_index < k)
  {
    shared_data[local_index] = means_x[local_index];
    shared_data[k + local_index] = means_y[local_index];
  }

  __syncthreads();

  // Load once here.
  const int32_t x_value = data_x[global_index];
  const int32_t y_value = data_y[global_index];

  int32_t best_distance = INT_MAX;
  int best_cluster = -1;
  for (int cluster = 0; cluster < k; ++cluster)
  {
    const int32_t distance = squared_l2_distance(x_value,
                                                y_value,
                                                shared_data[cluster],
                                                shared_data[k + cluster]);
    if (distance < best_distance)
    {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  __syncthreads();

  // reduction

  const int x = local_index;
  const int y = local_index + blockDim.x;
  const int count = local_index + blockDim.x + blockDim.x;

  for (int cluster = 0; cluster < k; ++cluster)
  {
    shared_data[x] = (best_cluster == cluster) ? x_value : 0;
    shared_data[y] = (best_cluster == cluster) ? y_value : 0;
    shared_data[count] = (best_cluster == cluster) ? 1 : 0;
    __syncthreads();

    // Reduction for this cluster.
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
      if (local_index < stride)
      {
        shared_data[x] += shared_data[x + stride];
        shared_data[y] += shared_data[y + stride];
        shared_data[count] += shared_data[count + stride];
      }
      __syncthreads();
    }

    // Now shared_data[0] holds the sum for x.

    if (local_index == 0)
    {
      const int cluster_index = blockIdx.x * k + cluster;
      new_sums_x[cluster_index] = shared_data[x];
      new_sums_y[cluster_index] = shared_data[y];
      counts[cluster_index] = shared_data[count];
    }
    __syncthreads();
  }
}

__global__ void coarse_reduce(int32_t *__restrict__ means_x,
                              int32_t *__restrict__ means_y,
                              int32_t *__restrict__ new_sum_x,
                              int32_t *__restrict__ new_sum_y,
                              int k,
                              int *__restrict__ counts)
{
  extern __shared__ int32_t shared_data[];

  const int index = threadIdx.x;
  const int y_offset = blockDim.x;

  shared_data[index] = new_sum_x[index];
  shared_data[y_offset + index] = new_sum_y[index];
  __syncthreads();

  for (int stride = blockDim.x / 2; stride >= k; stride /= 2)
  {
    if (index < stride)
    {
      shared_data[index] += shared_data[index + stride];
      shared_data[y_offset + index] += shared_data[y_offset + index + stride];
    }
    __syncthreads();
  }

  if (index < k)
  {
    const int count = max(1, counts[index]);
    means_x[index] = new_sum_x[index] / count;
    means_y[index] = new_sum_y[index] / count;
    new_sum_y[index] = 0;
    new_sum_x[index] = 0;
    counts[index] = 0;
  }
}

int main(int argc, char **argv)
{
  struct Params p = input_params(argc, argv);

  int k = p.k, number_of_elements = p.numPoints, number_of_iterations = p.maxItr;

  std::vector<int32_t> h_x (number_of_elements);
  std::vector<int32_t> h_y (number_of_elements);
  getVector(number_of_elements, h_x);
  getVector(number_of_elements, h_y);

  Data d_data(number_of_elements, h_x, h_y);

  std::mt19937 rng(std::random_device{}());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  Data d_means(k, h_x, h_y);

  const int threads = 1024;
  const int blocks = (number_of_elements + threads - 1) / threads;

  std::cout << "Processing " << number_of_elements << " points on " << blocks
            << " blocks x " << threads << " threads" << std::endl;

  // * 3 for x, y and counts.
  const int fine_shared_memory = 3 * threads * sizeof(int32_t);
  // * 2 for x and y. Will have k * blocks threads for the coarse reduction.
  const int coarse_shared_memory = 2 * k * blocks * sizeof(int32_t);

  hipError_t errorCode;

  Data d_sums(k * blocks);
  int *d_counts;
  errorCode = hipMalloc(&d_counts, k * blocks * sizeof(int));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemset(d_counts, 0, k * blocks * sizeof(int));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  std::cout << "Launching CUDA Kernel." << std::endl;

  auto [timeElapsed, avgPower, energy] = measureCUDAPowerAndElapsedTime([&]() {
    for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
      fine_reduce<<<blocks, threads, fine_shared_memory>>>(
          d_data.x, d_data.y, d_data.size, d_means.x, d_means.y, d_sums.x,
          d_sums.y, k, d_counts);
      errorCode = hipDeviceSynchronize();
      if (errorCode != hipSuccess) {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
      }

      coarse_reduce<<<1, k * blocks, coarse_shared_memory>>>(
          d_means.x, d_means.y, d_sums.x, d_sums.y, k, d_counts);

      errorCode = hipDeviceSynchronize();
      if (errorCode != hipSuccess) {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
      }
    }
  });

  printf("Execution time Kmeans = %f ms\n", timeElapsed);
  printf("Average Power = %f mW\n", avgPower);
  printf("Energy Consumption = %f mJ\n", energy);

  hipFree(d_counts);

  std::vector<int32_t> mean_x(k, 0);
  std::vector<int32_t> mean_y(k, 0);
  errorCode = hipMemcpy(mean_x.data(), d_means.x, d_means.bytes, hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(mean_y.data(), d_means.y, d_means.bytes, hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  for (size_t cluster = 0; cluster < k; ++cluster)
  {
    std::cout << mean_x[cluster] << " " << mean_y[cluster] << std::endl;
  }

  return 1;
}