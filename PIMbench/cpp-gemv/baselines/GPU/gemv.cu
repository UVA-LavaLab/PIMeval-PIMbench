#include "hip/hip_runtime.h"
/* File:     gemv.cu
 * Purpose:  Implement matrix vector multiplication on a gpu using cuda
 *
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hipblas.h>

#include "../../../utilBaselines.h"

#define TOLERANCE 200.0f

using namespace std;

vector<int32_t> A;
vector<int32_t> B;
vector<int32_t> C;

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t row, column;
  bool shouldVerify;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./gemv.out [options]"
          "\n"
          "\n    -r    row size (default=16384)"
          "\n    -c    column size (default=16384)"
          "\n    -v    t = verifies PIM output with host output. (default=false)"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.row = 16384;
  p.column = 16384;

  int opt;
  while ((opt = getopt(argc, argv, ":r:c:h:v:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'r':
      p.row = atoll(optarg);
      break;
    case 'c':
      p.column = atoll(optarg);
      break;
    case 'v':
      p.shouldVerify = (*optarg == 't') ? true : false;
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

int main(int argc, char *argv[])
{
  struct Params p = input_params(argc, argv);

  uint64_t row = p.row, col = p.column;
  getVector(row * col, A);
  getVector(col, B);
  C.resize(row);

  float *x, *y, *z;

  hipError_t errorCode;

  errorCode = hipMalloc((void **)&x, row * col * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc((void **)&y, col * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc((void **)&z, row * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(x, A.data(), row * col * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(y, B.data(), col * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  const float alpha = 1.0;
  const float beta = 0.0;
  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    std::cerr << "CUBLAS initialization failed\n";
    exit(1);
  }

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timeElapsed = 0;

  // Start timer
  hipEventRecord(start, 0);
  /* Kernel Call */
  status = hipblasSgemv(handle, HIPBLAS_OP_N, row, col, &alpha, x, row, y, 1, &beta, z, 1);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    std::cerr << "CUBLAS SGEMV failed\n";
    exit(1);
  }

  // Check for kernel launch errors
  errorCode = hipGetLastError();
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  // End timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timeElapsed, start, stop);

  printf("Execution time = %f ms\n", timeElapsed);

  errorCode = hipMemcpy(C.data(), z, row * sizeof(int32_t), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error Copy from host to device: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  if (p.shouldVerify)
  {
    cout.precision(0);
    for (int i = 0; i < row; ++i)
    {
      int32_t sum = 0;
      for (int j = 0; j < col; ++j)
      {
        sum += A[i + j * row] * B[j];
      }
      if (abs(C[i] - sum) > TOLERANCE)
      {
        cout << fixed << "Multiplication failed at index: " << i << "\t" << C[i] << "\t" << sum << endl;
        break;
      }
    }
    cout << "All correct!" << endl;
  }

  /* Free memory */
  hipblasDestroy(handle);
  hipFree(x);
  hipFree(y);
  hipFree(z);

  return 0;
} /* main */
