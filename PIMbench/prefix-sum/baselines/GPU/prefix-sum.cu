#include "hip/hip_runtime.h"
/* File:     prefix-sum.cu
 * Purpose:  Implement prefix sum on gpu using cuda
 *
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <nvml.h>

#include "utilBaselines.h"

using namespace std;

vector<int32_t> A;
vector<int32_t> C;

#define TOLERANCE	200.0f

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t vector_size;
} Params;

void usage()
{
  fprintf(stderr,
          "\n Usage:  ./prefix-sum.out [options]"
          "\n"
          "\n    -i    vector size (default=65536)"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.vector_size = 65536;

  int opt;
  while ((opt = getopt(argc, argv, ":h:i:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'i':
      p.vector_size = atoll(optarg);
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

int main(int argc, char *argv[])
{
  struct Params p = input_params(argc, argv);

  uint64_t vector_size = p.vector_size;
  getVector(vector_size, A);
  int *d_in, *d_out;
  hipError_t errorCode;

  errorCode = hipMalloc(&d_in, sizeof(int) * vector_size);
  if (errorCode != hipSuccess)
  {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
  errorCode = hipMalloc(&d_out, sizeof(int) * vector_size);
  if (errorCode != hipSuccess)
  {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  // Determine temporary device storage requirements
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                vector_size);

  errorCode = hipMalloc(&d_temp_storage, temp_storage_bytes);
  if (errorCode != hipSuccess)
  {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(d_in, A.data(), sizeof(int) * vector_size, hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  auto [timeElapsed, avgPower, energy] = measureCUDAPowerAndElapsedTime([&]() {
    /* Kernel Call */
    errorCode = hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                vector_size);
  });

  printf("Execution time for Prefix Sum = %f ms\n", timeElapsed);
  printf("Average Power = %f mW\n", avgPower);
  printf("Energy Consumption = %f mJ\n", energy);

  vector<int32_t> C(vector_size);
  errorCode = hipMemcpy(C.data(), d_out, vector_size * sizeof(int32_t), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error Copy from device to host: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  cout.precision(0);
  int sum = 0;
  for (size_t i = 0; i < A.size(); ++i)
  {
    sum += A[i];
    if (abs(C[i] - sum) > TOLERANCE)
    {
      cout << fixed << "Prefix Sum failed at index: " << i << "\t" << C[i] << "\t" << sum << endl;
      break;
    }
  }

  /* Free memory */
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp_storage);

  return 0;
} /* main */
