/* File:     aes.cu
 * Purpose:  Implement aes on a gpu using cuda
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <vector>
#include <random>
#include <getopt.h>
#include <hip/hip_runtime.h>
#define MEASUREMENT_TIMES (1 << 4) 

uint8_t ctx_key[32]; 
uint8_t ctx_enckey[32]; 
uint8_t ctx_deckey[32];

#define AES_BLOCK_SIZE 16
#define THREADS_PER_BLOCK 512
#define AES_KEY_BUFFER_SIZE 32

#define F(x)   (((x)<<1) ^ ((((x)>>7) & 1) * 0x1b))
#define FD(x)  (((x) >> 1) ^ (((x) & 1) ? 0x8d : 0))

// Function to compare two files
int compare_files(const char *file1, const char *file2);

// S table
__constant__ static const uint8_t sbox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5,
    0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
    0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc,
    0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a,
    0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
    0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b,
    0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
    0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
    0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17,
    0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88,
    0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
    0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9,
    0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6,
    0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
    0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94,
    0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68,
    0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

// inv S table
__constant__ static const uint8_t sboxinv[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38,
    0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87,
    0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d,
    0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2,
    0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16,
    0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda,
    0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a,
    0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02,
    0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea,
    0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85,
    0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89,
    0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20,
    0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31,
    0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d,
    0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0,
    0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26,
    0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
};

// Params 
typedef struct Params
{
    uint64_t inputSize;
    char *keyFile;
    char *inputFile;
    char *cipherFile;
    char *outputFile;
    bool shouldVerify;
} Params;

void usage() {
    fprintf(stderr,
        "\nUsage:  ./aes.out [options]"
        "\n"
        "\n    -l    input size (default=65536 bytes)"
        "\n    -k    key file containing AES key (default=generates key with random numbers)"
        "\n    -i    input file containing AES encrption input(default=generates input with random numbers)"
        "\n    -c    cipher file containing AES encryption output (default=./cipher.txt)"
        "\n    -o    output file containing AES decryption output (default=./output.txt)"
        "\n    -v    (true/false) validates if the input file and outputfile are the same. (default=false)"
        "\n");
}

struct Params getInputParams(int argc, char **argv) {
    struct Params p = {65536, NULL, NULL, "./cipher.txt", "./output.txt", false};
    int opt;

    while ((opt = getopt(argc, argv, "hl:k:i:c:o:v:")) >= 0) {
        switch (opt) {
            case 'h':
                usage();
                exit(0);
                break;
            case 'l':
                p.inputSize = strtoull(optarg, NULL, 0);
                break;
            case 'k':
                p.keyFile = optarg;
                break;
            case 'i':
                p.inputFile = optarg;
                break;
            case 'c':
                p.cipherFile = optarg;
                break;
            case 'o':
                p.outputFile = optarg;
                break;
            case 'v':
                p.shouldVerify = (*optarg == 't') ? true : false;
                break;
            default:
                fprintf(stderr, "\nERROR: Unrecognized option!\n");
                usage();
                exit(0);
        }
    }
    return p;
}

// x-time operation
__device__ uint8_t rj_xtime(uint8_t x){
  return (x & 0x80) ? ((x << 1) ^ 0x1b) : (x << 1);
}

// subbyte operation
__device__ void aes_subBytes(uint8_t *buf){
  register uint8_t i, b;
  for (i = 0; i < 16; ++i){
    b = buf[i];
    buf[i] = sbox[b];
  }
} 

// inv subbyte operation
__device__ void aes_subBytes_inv(uint8_t *buf){
  register uint8_t i, b;
  for (i = 0; i < 16; ++i){
    b = buf[i];
    buf[i] = sboxinv[b];
  }
} 

// add round key operation
__device__ void aes_addRoundKey(uint8_t *buf, uint8_t *key){
  register uint8_t i = 16;
  while (i--){
    buf[i] ^= key[i];
  }
} 

// add round key at beginning
__device__ void aes_addRoundKey_cpy(uint8_t *buf, uint8_t *key, uint8_t *cpk){
  register uint8_t i = 16;
  while (i--){
    buf[i] ^= (cpk[i] = key[i]);
    cpk[16+i] = key[16 + i];
  }
} 

// shift row operation
__device__ void aes_shiftRows(uint8_t *buf){
  register uint8_t i, j; 
  i = buf[1];
  buf[1] = buf[5];
  buf[5] = buf[9];
  buf[9] = buf[13];
  buf[13] = i;
  i = buf[10];
  buf[10] = buf[2];
  buf[2] = i;
  j = buf[3];
  buf[3] = buf[15];
  buf[15] = buf[11];
  buf[11] = buf[7];
  buf[7] = j;
  j = buf[14];
  buf[14] = buf[6];
  buf[6]  = j;
}

// inv shift row operation
__device__ void aes_shiftRows_inv(uint8_t *buf){
  register uint8_t i, j; 
  i = buf[1];
  buf[1] = buf[13];
  buf[13] = buf[9];
  buf[9] = buf[5];
  buf[5] = i;
  i = buf[2];
  buf[2] = buf[10];
  buf[10] = i;
  j = buf[3];
  buf[3] = buf[7];
  buf[7] = buf[11];
  buf[11] = buf[15];
  buf[15] = j;
  j = buf[6];
  buf[6] = buf[14];
  buf[14] = j;
} 

// mix column operation
__device__ void aes_mixColumns(uint8_t *buf){
  register uint8_t i, a, b, c, d, e;
  for (i = 0; i < 16; i += 4){
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    buf[i] ^= e ^ rj_xtime(a^b);
    buf[i+1] ^= e ^ rj_xtime(b^c);
    buf[i+2] ^= e ^ rj_xtime(c^d);
    buf[i+3] ^= e ^ rj_xtime(d^a);
  }
} 

// inv mix column operation
__device__ void aes_mixColumns_inv(uint8_t *buf){
  register uint8_t i, a, b, c, d, e, x, y, z;
  for (i = 0; i < 16; i += 4){
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    z = rj_xtime(e);
    x = e ^ rj_xtime(rj_xtime(z^a^c));
    y = e ^ rj_xtime(rj_xtime(z^b^d));
    buf[i] ^= x ^ rj_xtime(a^b);
    buf[i+1] ^= y ^ rj_xtime(b^c);
    buf[i+2] ^= x ^ rj_xtime(c^d);
    buf[i+3] ^= y ^ rj_xtime(d^a);
  }
} 

// add expand key operation
__device__ __host__ void aes_expandEncKey(uint8_t *k, uint8_t *rc, const uint8_t *sb){
  register uint8_t i;

  k[0] ^= sb[k[29]] ^ (*rc);
  k[1] ^= sb[k[30]];
  k[2] ^= sb[k[31]];
  k[3] ^= sb[k[28]];
  *rc = F( *rc);

  for(i = 4; i < AES_BLOCK_SIZE; i += 4){
    k[i] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

  k[16] ^= sb[k[12]];
  k[17] ^= sb[k[13]];
  k[18] ^= sb[k[14]];
  k[19] ^= sb[k[15]];

  for(i = 20; i < AES_KEY_BUFFER_SIZE; i += 4){
    k[i] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

} 

// inv add expand key operation
__device__ void aes_expandDecKey(uint8_t *k, uint8_t *rc){
  uint8_t i;

  for(i = 28; i > 16; i -= 4){
    k[i+0] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

  k[16] ^= sbox[k[12]];
  k[17] ^= sbox[k[13]];
  k[18] ^= sbox[k[14]];
  k[19] ^= sbox[k[15]];

  for(i = 12; i > 0; i -= 4){
    k[i+0] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

  *rc = FD(*rc);
  k[0] ^= sbox[k[29]] ^ (*rc);
  k[1] ^= sbox[k[30]];
  k[2] ^= sbox[k[31]];
  k[3] ^= sbox[k[28]];
} 


// key initition
void aes256_init(uint8_t *k){
  uint8_t rcon = 1;
  register uint8_t i;

  for (i = 0; i < sizeof(ctx_key); i++){
    ctx_enckey[i] = ctx_deckey[i] = k[i];
  }
  for (i = 8;--i;){
    aes_expandEncKey(ctx_deckey, &rcon, sbox);
  }
} 

// aes encrypt algorithm one thread/one block with AES_BLOCK_SIZE 
__global__ void aes256_encrypt_ecb(uint8_t *buf_d, unsigned long numbytes, uint8_t *ctx_enckey_d, uint8_t *ctx_key_d){
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE]; // thread buffer
  //printf("Thread %d\n", threadIdx.x);
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) {  return; }

  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_enckey_d, ctx_key_d);
  for(i = 1, rcon = 1; i < 14; ++i){
    aes_subBytes(buf_t);
    aes_shiftRows(buf_t);
    aes_mixColumns(buf_t);
    if( i & 1 ){
      aes_addRoundKey( buf_t, &ctx_key_d[16]);
    }
    else{
      aes_expandEncKey(ctx_key_d, &rcon, sbox), aes_addRoundKey(buf_t, ctx_key_d);
    }
  }
  aes_subBytes(buf_t);
  aes_shiftRows(buf_t);
  aes_expandEncKey(ctx_key_d, &rcon, sbox);
  aes_addRoundKey(buf_t, ctx_key_d);
  /* copy thread buffer back into global memory */
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} 

// aes decrypt algorithm
__global__ void aes256_decrypt_ecb(uint8_t *buf_d, unsigned long numbytes, uint8_t *ctx_deckey_d, uint8_t *ctx_key_d){
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE];
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) { return; }
  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_deckey_d, ctx_key_d);
  aes_shiftRows_inv(buf_t);
  aes_subBytes_inv(buf_t);
  for (i = 14, rcon = 0x80; --i;){
    if( ( i & 1 ) ){
      aes_expandDecKey(ctx_key_d, &rcon);
      aes_addRoundKey(buf_t, &ctx_key_d[16]);
    }
    else{
      aes_addRoundKey(buf_t, ctx_key_d);
    }
    aes_mixColumns_inv(buf_t);
    aes_shiftRows_inv(buf_t);
    aes_subBytes_inv(buf_t);
    }
  aes_addRoundKey( buf_t, ctx_key_d);
  /* copy thread back into global memory */
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} 

// aes encrypt demo
void encryptdemo(uint8_t key[AES_KEY_BUFFER_SIZE], uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;
  uint8_t *ctx_key_d, *ctx_enckey_d;
  hipError_t errorCode;

  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(sbox), sbox, sizeof(uint8_t)*256);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMalloc((void**)&buf_d, numbytes);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
  errorCode = hipMalloc((void**)&ctx_enckey_d, sizeof(ctx_enckey));
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
  errorCode = hipMalloc((void**)&ctx_key_d, sizeof(ctx_key));
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
 
  errorCode = hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_enckey_d, ctx_enckey, sizeof(ctx_enckey), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  aes256_encrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes, ctx_enckey_d, ctx_key_d);

  errorCode = hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_enckey, ctx_enckey_d, sizeof(ctx_enckey), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  hipFree(buf_d);
  hipFree(ctx_key_d);
  hipFree(ctx_enckey_d);
}

// aes decrypt demo
void decryptdemo(uint8_t key[AES_KEY_BUFFER_SIZE], uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;
  uint8_t *ctx_key_d, *ctx_deckey_d;
  hipError_t errorCode;

  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(sboxinv), sboxinv, sizeof(uint8_t)*256);

  hipMalloc((void**)&buf_d, numbytes);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
  hipMalloc((void**)&ctx_deckey_d, sizeof(ctx_deckey));
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }
  hipMalloc((void**)&ctx_key_d, sizeof(ctx_key));
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_deckey_d, ctx_deckey, sizeof(ctx_deckey), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  aes256_decrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes, ctx_deckey_d, ctx_key_d);

  errorCode = hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_deckey, ctx_deckey_d, sizeof(ctx_deckey), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }

  errorCode = hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
      exit(1);
  }


  hipFree(buf_d);
  hipFree(ctx_key_d);
  hipFree(ctx_deckey_d);
}

__global__ void GPU_init() { }

// Tester function
void test_encryptdemo_parallel() {
    const unsigned long SIZE = 1UL * 1024 * 1024 * 1024; // 1 GB
    uint8_t key[AES_KEY_BUFFER_SIZE];

    // Fill the key with random bytes
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);
    for (auto& k : key) {
        k = dis(gen);
    }

    // Allocate 1GB buffer and fill with random data
    std::vector<uint8_t> buffer(SIZE);
    for (auto& byte : buffer) {
        byte = dis(gen);
    }

    // This is to force nvcc to put the GPU initialization here.
    GPU_init<<<1, 1>>>();
  
    aes256_init(key);

    // Start encrypt in CPU
    auto start = std::chrono::high_resolution_clock::now();
    for (int k = 0; k < MEASUREMENT_TIMES; k++) {
        encryptdemo(key, buffer.data(), SIZE);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsedTime = (end - start) / MEASUREMENT_TIMES;
    std::cout << "Encryption Duration: " << std::fixed << std::setprecision(6) << elapsedTime.count() << " ms." << std::endl;

}

void test_decryptdemo_parallel() {
    const unsigned long SIZE = 1UL * 1024 * 1024 * 1024; // 1 GB
    uint8_t key[AES_KEY_BUFFER_SIZE];

    // Fill the key with random bytes
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);
    for (auto& k : key) {
        k = dis(gen);
    }

    // Allocate 1GB buffer and fill with random data
    std::vector<uint8_t> buffer(SIZE);
    for (auto& byte : buffer) {
        byte = dis(gen);
    }

    // This is to force nvcc to put the GPU initialization here.
    GPU_init<<<1, 1>>>();
  
    aes256_init(key);


    // Start encrypt in CPU
    auto start = std::chrono::high_resolution_clock::now();
    for (int k = 0; k < MEASUREMENT_TIMES; k++) {
        decryptdemo(key, buffer.data(), SIZE);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsedTime = (end - start) / MEASUREMENT_TIMES;
    std::cout << "Decryption Duration: " << std::fixed << std::setprecision(6) << elapsedTime.count() << " ms." << std::endl;

}

int main(int argc, char *argv[]) {
    struct Params params = getInputParams(argc, argv);
    
    FILE *file; uint8_t *buf; 
    int padding;
    uint8_t key[AES_KEY_BUFFER_SIZE];
    unsigned long long numbytes; 
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error_id));
        printf("Exiting...\n");
        return EXIT_FAILURE;
    }

    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
        return EXIT_FAILURE;
    }

    if (params.keyFile == NULL) {
        printf("INFO: Key file is not specifed. Random key will be used.\n");
        for (unsigned int i = 0; i < AES_KEY_BUFFER_SIZE; ++i) {
            key[i] = rand() & 0xff;
        }
    } else {
        // Open and read the key file.
        file = fopen(params.keyFile, "r");
        if (file == NULL) {
            printf("ERROR: Error opening key file %s\n", params.keyFile);
            return EXIT_FAILURE;
        }
        if (fread(key, 1, AES_KEY_BUFFER_SIZE, file) != AES_KEY_BUFFER_SIZE) {
          printf("ERROR: The key length in %s is not %d characters\n", params.keyFile, AES_KEY_BUFFER_SIZE);
          fclose(file);
          return EXIT_FAILURE;
        } 
        // Verify that there are no extra characters.
        char extra;
        if (fread(&extra, 1, 1, file) != 0) {
            printf("ERROR: The key length in %s is more than %d characters\n", params.keyFile, AES_KEY_BUFFER_SIZE);
            fclose(file);
            return EXIT_FAILURE;
        }
        fclose(file);
    }
   
    // Allocate memory for the file content.
    numbytes = params.inputSize;
    buf = (uint8_t*)calloc(numbytes, sizeof(uint8_t));
    if (buf == NULL) {
        printf("ERROR: Memory allocation error\n");
        return EXIT_FAILURE;
    }
 
    // Open and read the input file.
    if (params.inputFile == NULL) {
        printf("INFO: Input file is not specifed. Random input will be used.\n");
        for (unsigned int i = 0; i < params.inputSize; ++i) {
            buf[i] = rand() & 0xff;
        }
    } else {
        file = fopen(params.inputFile, "r");
        if (file == NULL) {
            printf("ERROR: Error opening input file %s\n", params.inputFile);
            free(buf);
            return EXIT_FAILURE;
        } 
        fseek(file, 0L, SEEK_END);
        numbytes = ftell(file);
        fseek(file, 0L, SEEK_SET);

        // Read the file into the buffer.
        if (fread(buf, 1, numbytes, file) != numbytes) {
            printf("ERROR: Unable to read all bytes from file %s\n", params.inputFile);
            fclose(file);
            free(buf);
            return EXIT_FAILURE;
        }
        fclose(file);
    }

    // Calculate padding.
    padding = AES_BLOCK_SIZE - (numbytes % AES_BLOCK_SIZE);
    numbytes += padding;
    printf("Padding file with %d bytes for a new size of %lu\n", padding, numbytes);

    // This is to force nvcc to put the GPU initialization here.
    GPU_init<<<1, 1>>>();
  
    aes256_init(key);
    // Encryption.
    auto start = std::chrono::high_resolution_clock::now();
    for (int k = 0; k < MEASUREMENT_TIMES; k++) {
        encryptdemo(key, buf, numbytes);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsedTime = (end - start) / MEASUREMENT_TIMES;
    std::cout << "Encryption Duration: " << std::fixed << std::setprecision(3) << elapsedTime.count() << " ms." << std::endl;
    std::cout << "GPU encryption throughput: " << std::fixed << std::setprecision(3) << (numbytes / elapsedTime.count() * 1000) << " bytes/second\n";

    // Write the ciphertext to file
    file = fopen(params.cipherFile, "w");
    if (file == NULL) {
        printf("ERROR: Error opening cipher file %s\n", params.cipherFile);
        free(buf);
        return EXIT_FAILURE;
    } 
    fwrite(buf, 1, numbytes, file);
    fclose(file);

    // Decryption.
    start = std::chrono::high_resolution_clock::now();
    for (int k = 0; k < MEASUREMENT_TIMES; k++) {
        decryptdemo(key, buf, numbytes);
    }
    end = std::chrono::high_resolution_clock::now();
    elapsedTime = (end - start) / MEASUREMENT_TIMES;
    std::cout << "Decryption Duration: " << std::fixed << std::setprecision(3) << elapsedTime.count() << " ms." << std::endl;
    std::cout << "GPU decryption throughput: " << std::fixed << std::setprecision(3) << (numbytes / elapsedTime.count() * 1000) << " bytes/second\n";

    // Write to output file
    file = fopen(params.outputFile, "w");
    if (file == NULL) {
        printf("ERROR: Error opening output file %s\n", params.outputFile);
        free(buf);
        return EXIT_FAILURE;
    } 
    fwrite(buf, 1, numbytes - padding, file);
    fclose(file);

    // Compare input and output files
    if (params.shouldVerify) { 
        if (compare_files(params.inputFile, params.outputFile) == 0) {
            printf("INFO: The input file and the output file are the same.\n");
        } else {
            printf("ERROR: The input file and the output file are different.\n");
        }
    }

    free(buf);
    return EXIT_SUCCESS;
}

int compare_files(const char *file1, const char *file2) {
    FILE *f1 = fopen(file1, "r");
    FILE *f2 = fopen(file2, "r");
    if (f1 == NULL || f2 == NULL) {
        if (f1) fclose(f1);
        if (f2) fclose(f2);
        return -1;
    }

    int ch1, ch2;
    do {
        ch1 = fgetc(f1);
        ch2 = fgetc(f2);
        if (ch1 != ch2) {
            fclose(f1);
            fclose(f2);
            return -1;
        }
    } while (ch1 != EOF && ch2 != EOF);

    fclose(f1);
    fclose(f2);

    if (ch1 == EOF && ch2 == EOF) {
        return 0;
    } else {
        return -1;
    }
}
