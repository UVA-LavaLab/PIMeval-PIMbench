#include "hip/hip_runtime.h"
/* File:     axpy.cu
 * Purpose:  Implement  on a gpu using cuda
 *
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hipblas.h>

#include "../../../utilBaselines.h"

using namespace std;

vector<int32_t> A;
vector<int32_t> B;
vector<int32_t> C;

#define TOLERANCE	200.0f

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t vector_size;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./axpy.out [options]"
          "\n"
          "\n    -i    vector size (default=65536)"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.vector_size = 65536;

  int opt;
  while ((opt = getopt(argc, argv, ":h:i:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'i':
      p.vector_size = atoll(optarg);
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

int main(int argc, char *argv[])
{
  struct Params p = input_params(argc, argv);

  uint64_t vector_size = p.vector_size;
  getVector(vector_size, A);
  getVector(vector_size, B);
  const float a = rand() % 5;

  float *x, *y;

  hipError_t errorCode;

  errorCode = hipMalloc((void **)&x, vector_size * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc((void **)&y, vector_size * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(x, A.data(), vector_size * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(y, B.data(), vector_size * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    std::cerr << "CUBLAS initialization failed\n";
    exit(1);
  }

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timeElapsed = 0;

  // Start timer
  hipEventRecord(start, 0);
  /* Kernel Call */
  status = hipblasSaxpy(handle, vector_size, &a, x, 1, y, 1);

  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    std::cerr << "CUBLAS SGEMV failed\n";
    exit(1);
  }

  // Check for kernel launch errors
  errorCode = hipGetLastError();
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  // End timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timeElapsed, start, stop);

  printf("Execution time = %f ms\n", timeElapsed);

  vector<int32_t> C(vector_size);
  errorCode = hipMemcpy(C.data(), y, vector_size * sizeof(int32_t), hipMemcpyDeviceToHost);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error Copy from device to host: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  cout.precision(0);
  for (size_t i = 0; i < A.size(); ++i)
  {
    int32_t sum = a * B[i] + A[i];
    if (abs(C[i] - sum) > TOLERANCE)
    {
      cout << fixed << "AXPY failed at index: " << i << "\t" << C[i] << "\t" << sum << endl;
      break;
    }
  }

  /* Free memory */
  hipblasDestroy(handle);
  hipFree(x);
  hipFree(y);

  return 0;
} /* main */
