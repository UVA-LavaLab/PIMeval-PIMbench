/* File:     matrix matrix multiplication cuda
 * Purpose:  Implement  on a gpu using cuda
 *
 */
//TODO: support different data type

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hipblas.h>

#include "../../../utilBaselines.h"

using namespace std;

#define TOLERANCE	200.0f

vector<int32_t> A;
vector<int32_t> B;
vector<int32_t> C;

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t row, column_A, column_B;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./gemv [options]"
          "\n"
          "\n    -r <R>    row size"
          "\n    -c <C>    MatA column size"
          "\n    -d <C>    MatB column size"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.row = 1024;
  p.column_A = 1024;
  p.column_B = 1024;

  int opt;
  while ((opt = getopt(argc, argv, ":r:c:d:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'r':
      p.row = atoll(optarg);
      break;
    case 'c':
      p.column_A = atoll(optarg);
      break;
    case 'd':
      p.column_B = atoll(optarg);
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

int main(int argc, char *argv[])
{
  struct Params p = input_params(argc, argv);

  uint64_t row = p.row, col_A = p.column_A, col_B = p.column_B;

  getVector(row * col_A, A);
  getVector(col_A * col_B, B);

  float *x, *y, *z;

  hipError_t errorCode;

  errorCode = hipMalloc(&x, row * col_A * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc(&y, col_A * col_B * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc(&z, row * col_B * sizeof(int32_t));
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(x, A.data(), row * col_A * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(y, B.data(), col_A * col_B * sizeof(float), hipMemcpyHostToDevice);
  if (errorCode != hipSuccess)
  {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  const float alpha = 1.0;
  const float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timeElapsed = 0;

  // Start timer
  hipEventRecord(start, 0);
  /* Kernel Call */
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col_B, col_A, &alpha, x, row, y, col_A, &beta, z, row);

  // End timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timeElapsed, start, stop);

  printf("Execution time = %f ms\n", timeElapsed);

  /* Free memory */
  hipFree(x);
  hipFree(y);
  hipFree(z);

  return 0;
} /* main */
