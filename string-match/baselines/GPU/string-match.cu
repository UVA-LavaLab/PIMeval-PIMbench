// Test: Cuda version of string match
// Copyright (c) 2025 University of Virginia
// This file is licensed under the MIT License.
// See the LICENSE file in the root of this repository for more details.

#include <iostream>
#include <vector>
#include <getopt.h>
#include <stdint.h>
#include <iomanip>
#include <cassert>
#if defined(_OPENMP)
#include <omp.h>
#endif

#include "utilStringMatch.h"
#include "PFAC.h"

// Params ---------------------------------------------------------------------
typedef struct Params
{
  const char *keysInputFile;
  const char *textInputFile;
  bool shouldVerify;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./string-match.out [options]"
          "\n"
          "\n    -k    keys input file, each key on new line (default=dataset/10mil_l-10_nk-10_kl/keys.txt) must be sorted by increasing length, must have a blank line at end of file"
          "\n    -t    text input file to search for keys from (default=dataset/10mil_l-10_nk-10_kl/text.txt)"
          "\n    -v    t = verifies PIM output with host output. (default=false)"
          "\n");
}

struct Params getInputParams(int argc, char **argv)
{
  struct Params p;
  p.keysInputFile = "./../../dataset/10mil_l-10_nk-10_kl/keys.txt";
  p.textInputFile = "./../../dataset/10mil_l-10_nk-10_kl/text.txt";
  p.shouldVerify = false;

  int opt;
  while ((opt = getopt(argc, argv, "h:k:t:v:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'k':
      p.keysInputFile = optarg;
      break;
    case 't':
      p.textInputFile = optarg;
      break;
    case 'v':
      p.shouldVerify = (*optarg == 't') ? true : false;
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

float stringMatchGpu(const char* needlesFilename, std::string& haystack, std::vector<int>& matches) {
  PFAC_handle_t pfacHandle;
  PFAC_status_t pfacError;
  hipError_t hipError_t;

  pfacError = PFAC_create(&pfacHandle);
  if ( PFAC_STATUS_SUCCESS != pfacError ){
      std::cerr << "Pfac Error: " << PFAC_getErrorString(pfacError) << std::endl;
      exit(1);
  }

  pfacError = PFAC_setPlatform(pfacHandle, PFAC_PLATFORM_GPU);
  if ( PFAC_STATUS_SUCCESS != pfacError ){
      std::cerr << "Pfac Error: " << PFAC_getErrorString(pfacError) << std::endl;
      exit(1);
  }

  pfacError = PFAC_readPatternFromFile(pfacHandle, needlesFilename);
  if (PFAC_STATUS_SUCCESS != pfacError){
    std::cerr << "Cuda Error: " << PFAC_getErrorString(pfacError) << std::endl;
    exit(1);
  }

  pfacError = PFAC_setTextureMode(pfacHandle, PFAC_TEXTURE_ON);
  if ( PFAC_STATUS_SUCCESS != pfacError ){
      std::cerr << "Pfac Error: " << PFAC_getErrorString(pfacError) << std::endl;
      exit(1);
  }

  char *gpuText;
  int *gpuMatches;

  size_t cudaToAlloc = (haystack.size() + sizeof(int)-1)/sizeof(int);
  hipError_t = hipMalloc((void **) &gpuText, cudaToAlloc*sizeof(int));
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }
  
  hipError_t = hipMalloc((void **) &gpuMatches, haystack.size()*sizeof(int));
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }
  
  hipError_t = hipMemcpy(gpuText, haystack.c_str(), haystack.size(), hipMemcpyHostToDevice);
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  float timeElapsed = 0;

  pfacError = PFAC_matchFromDevice(pfacHandle, gpuText, haystack.size(), gpuMatches, &timeElapsed);
  if (PFAC_STATUS_SUCCESS != pfacError){
      std::cerr << "Pfac Error: " << PFAC_getErrorString(pfacError) << std::endl;
      exit(1);
  }

  hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  pfacError = PFAC_destroy(pfacHandle);
  if ( PFAC_STATUS_SUCCESS != pfacError ){
      std::cerr << "Pfac Error: " << PFAC_getErrorString(pfacError) << std::endl;
      exit(1);
  }

  hipError_t = hipMemcpy(matches.data(), gpuMatches, haystack.size() * sizeof(int), hipMemcpyDeviceToHost);
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t = hipFree(gpuText);
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t = hipFree(gpuMatches);
  if(hipError_t != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  return timeElapsed;
}

int main(int argc, char* argv[])
{
  struct Params params = getInputParams(argc, argv);
  
  std::cout << "Running GPU string match for \"" << params.keysInputFile << "\" as the keys file, and \"" << params.textInputFile << "\" as the text input file\n";
  
  std::string haystack;
  std::vector<std::string> needles;
  std::vector<int> matches;

  haystack = readStringFromFile(params.textInputFile);
  needles = getNeedlesFromFile(params.keysInputFile);

  matches.resize(haystack.size());

  float timeElapsed = stringMatchGpu(params.keysInputFile, haystack, matches);
  printf("Execution time of string match = %f ms\n", timeElapsed);

  if (params.shouldVerify) 
  {
    std::vector<int> matchesCpu;
    
    matchesCpu.resize(haystack.size(), 0);

    stringMatchCpu(needles, haystack, matchesCpu);

    // verify result
    bool ok = true;
    #pragma omp parallel for
    for (unsigned i = 0; i < matches.size(); ++i)
    {
      if (matches[i] != matchesCpu[i])
      {
        std::cout << "Wrong answer: " << unsigned(matches[i]) << " (expected " << unsigned(matchesCpu[i]) << "), for position " << i << std::endl;
        ok = false;
      }
    }
    if(ok) {
      std::cout << "Correct for string match!" << std::endl;
    }
  }

  return 0;
}
